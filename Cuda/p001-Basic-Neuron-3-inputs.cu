
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

int main() {
  float *inputs, *weights, *bias, *output;

  hipMallocManaged(&inputs, 3*sizeof(float));
  hipMallocManaged(&weights, 3*sizeof(float));
  hipMallocManaged(&bias, sizeof(float));
  hipMallocManaged(&output, sizeof(float));

  inputs[0] = 1.0f;
  inputs[1] = 2.0f;
  inputs[2] = 3.0f;

  weights[0] = 3.1f;
  weights[1] = 2.1f;
  weights[2] = 8.7f;

  bias[0] = 3.0f;

  output[0] = inputs[0] * weights[0] + inputs[1] * weights[1] +
                 inputs[2] * weights[2] + bias[0];

  std::cout << output[0] << std::endl;

  hipFree(inputs);
  hipFree(weights);
  hipFree(bias);
  hipFree(output);
}
