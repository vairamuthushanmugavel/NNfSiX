
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define NUM_INPUTS 4
#define NUM_OUTPUTS 3

int main() {
  float *inputs, *weights1, *bias1, *weights2, *bias2, *weights3, *bias3, *output;

  hipMallocManaged(&inputs, NUM_INPUTS*sizeof(float));
  hipMallocManaged(&weights1, NUM_INPUTS*sizeof(float));
  hipMallocManaged(&bias1, sizeof(float));
  hipMallocManaged(&weights2, NUM_INPUTS*sizeof(float));
  hipMallocManaged(&bias2, sizeof(float));
  hipMallocManaged(&weights3, NUM_INPUTS*sizeof(float));
  hipMallocManaged(&bias3, sizeof(float));
  hipMallocManaged(&output, NUM_OUTPUTS*sizeof(float));

  inputs[0] = 1.0f;
  inputs[1] = 2.0f;
  inputs[2] = 3.0f;
  inputs[3] = 2.5f;

  weights1[0] = 0.2f;
  weights1[1] = 0.8f;
  weights1[2] = -0.5f;
  weights1[3] = 1.0f;

  weights2[0] = 0.5f;
  weights2[1] = -0.91f;
  weights2[2] = 0.26f;
  weights2[3] = -0.5f;

  weights3[0] = -0.26f;
  weights3[1] = -0.27f;
  weights3[2] = 0.17f;
  weights3[3] = 0.87f;

  bias1[0] = 2.0f;
  bias2[0] = 3.0f;
  bias3[0] = 0.5f;

  output[0] = inputs[0]*weights1[0] + inputs[1]*weights1[1] + inputs[2]*weights1[2] + inputs[3]*weights1[3] + bias1[0];
  output[1] = inputs[0]*weights2[0] + inputs[1]*weights2[1] + inputs[2]*weights2[2] + inputs[3]*weights2[3] + bias2[0];
  output[2] = inputs[0]*weights3[0] + inputs[1]*weights3[1] + inputs[2]*weights3[2] + inputs[3]*weights3[3] + bias3[0];

  std::cout << "[" << output[0] << ", " << output[1] << ", " << output[2] << "]" << std::endl;

  hipFree(inputs);
  hipFree(weights1);
  hipFree(bias1);
  hipFree(weights2);
  hipFree(bias2);
  hipFree(weights3);
  hipFree(bias3);
  hipFree(output);
}
