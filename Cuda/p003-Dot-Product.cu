
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define NUM_INPUTS 4
#define NUM_OUTPUTS 3

__global__
void dot(int outputs, int inputs, float *M, float *v, float *u){
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int j = index; j < outputs; j += stride){
    u[j] = 0.0f;
    for(int i = 0; i < inputs; i++){
      u[j] += M[i + inputs*j]*v[i];
    }
  }
}

__global__
void add(int n, float *u, float *v){
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride){
    u[i] = u[i] + v[i];
  }
}

int main() {
  float *inputs, *weights, *bias, *output;

  hipMallocManaged(&inputs, NUM_INPUTS*sizeof(float));
  hipMallocManaged(&weights, NUM_INPUTS*NUM_OUTPUTS*sizeof(float));
  hipMallocManaged(&bias, NUM_OUTPUTS*sizeof(float));
  hipMallocManaged(&output, NUM_OUTPUTS*sizeof(float));

  inputs[0] = 1.0f;
  inputs[1] = 2.0f;
  inputs[2] = 3.0f;
  inputs[3] = 2.5f;

  weights[0 + NUM_INPUTS*0] = 0.2f; //indexing as [x, y] = [x + NUM_INPUTS*y]
  weights[1 + NUM_INPUTS*0] = 0.8f;
  weights[2 + NUM_INPUTS*0] = -0.5f;
  weights[3 + NUM_INPUTS*0] = 1.0f;

  weights[0 + NUM_INPUTS*1] = 0.5f;
  weights[1 + NUM_INPUTS*1] = -0.91f;
  weights[2 + NUM_INPUTS*1] = 0.26f;
  weights[3 + NUM_INPUTS*1] = -0.5f;

  weights[0 + NUM_INPUTS*2] = -0.26f;
  weights[1 + NUM_INPUTS*2] = -0.27f;
  weights[2 + NUM_INPUTS*2] = 0.17f;
  weights[3 + NUM_INPUTS*2] = 0.87f;

  bias[0] = 2.0f;
  bias[1] = 3.0f;
  bias[2] = 0.5f;

  dot<<<1, NUM_OUTPUTS>>>(NUM_OUTPUTS, NUM_INPUTS, weights, inputs, output);
  add<<<1, NUM_OUTPUTS>>>(NUM_OUTPUTS, output, bias);
  hipDeviceSynchronize();


  std::cout << "[" << output[0] << ", " << output[1] << ", " << output[2] << "]" << std::endl;

  hipFree(inputs);
  hipFree(weights);
  hipFree(bias);
  hipFree(output);
}
